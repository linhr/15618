#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "matrix.h"
#include "cuda_algebra.h"
#include "cycle_timer.h"

#define WARP_SIZE 32
#define THREADS_PER_BLOCK 512

using std::vector;

/**
 * @brief   Cuda kernel function for vector copy.
 *
 * @param   N   The vector size.
 * @param   y   The dest vector.
 * @param   x   The src vector.
 */
template <typename T>
__global__ void vector_copy_kernel(const int N, T *y, const T *x) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        y[index] = x[index];
    }
}

/**
 * @brief   Cuda kernel function for vector dot product.
 *
 * @param   N   The vector size.
 * @param   x   The first input vector.
 * @param   y   The second input vector.
 * @param   z   The temp sum per block.
 */
template <typename T>
__global__ void dot_product_kernel(const int N, const T *x, const T *y,
    T *z) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ T result[THREADS_PER_BLOCK];

    if (index < N) {
        result[threadIdx.x] = x[index] * y[index];
    } else {
        result[threadIdx.x] = 0;
    }
    __syncthreads();

    int half = THREADS_PER_BLOCK / 2;
    while (half > 0) {
        if (threadIdx.x < half) {
            result[threadIdx.x] += result[threadIdx.x + half];
        }
        __syncthreads();
        half /= 2;
    }

    if (threadIdx.x == 0) {
        z[blockIdx.x] = result[0];
    }
}

/**
 * @brief   Cuda kernel function for vector multiply in place.
 *
 * @param   N   The vector size.
 * @param   x   The input vector.
 * @param   k   The value to multiply.
 */
template <typename T>
__global__ void multiply_inplace_kernel(const int N, T *x, const T k) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        x[index] = x[index] * k;
    }
}

/**
 * @brief   Cuda kernel function for vector add in place.
 *
 * @param   N   The vector size.
 * @param   x   The input vector.
 * @param   k   The value to add.
 */
template <typename T>
__global__ void add_inplace_kernel(const int N, T *x, const T k) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        x[index] = x[index] + k;
    }
}

/**
 * @brief   Cuda kernel function for vector-vector add in place.
 *
 * @param   N   The vector size.
 * @param   x   The input vector.
 * @param   y   The other vector to add.
 */
template <typename T>
__global__ void vec_add_inplace_kernel(const int N, T *x, const T *y) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        x[index] = x[index] + y[index];
    }
}

/**
 * @brief   Cuda kernel function for vector saxpy in place(y += a * x).
 *
 * @param   N   The vector size.
 * @param   y   The output vector.
 * @param   x   The input vector.
 * @param   a   The value to multiply.
 */
template <typename T>
__global__ void saxpy_inplace_kernel(const int N, T *y, const T *x,
    const T a) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) {
        y[index] += a * x[index];
    }
}

/**
 * @brief   Cuda kernel function for naive sparse matrix multiplication.
 *
 * @param   rows    The row number of the matrix.
 * @param   row_ptr Row pointers in the CSR matrix.
 * @param   col_ind Column indexes in the CSR matrix.
 * @param   values  Data values in the CSR matrix.
 * @param   x       The input vector x to multiply.
 * @param   y       The output vector y.
 */
template <typename T>
__global__ void naive_multiply_kernel(const int rows, const int *row_ptr,
    const int *col_ind, const T *values, const T *x, T *y) {

    int r = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < rows) {
        T result(0);
        int start = row_ptr[r];
        int end = row_ptr[r + 1];
        for (int i = start; i < end; i++) {
            result += values[i] * x[col_ind[i]];
        }
        y[r] = result;
    }
}

/**
 * @brief   Cuda kernel function for naive sparse matrix multiplication.
 *
 * @param   rows    The row number of the matrix.
 * @param   row_ptr Row pointers in the CSR matrix.
 * @param   col_ind Column indexes in the CSR matrix.
 * @param   values  Data values in the CSR matrix.
 * @param   x       The input vector x to multiply.
 * @param   y       The output vector y.
 */
template <typename T>
__global__ void warp_multiply_kernel(const int rows, const int *row_ptr,
    const int *col_ind, const T *values, const T *x, T *y) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int r = index / WARP_SIZE;
    int lane = index % WARP_SIZE;
    __shared__ T result[THREADS_PER_BLOCK];

    if (r < rows) {
        int start = row_ptr[r];
        int end = row_ptr[r + 1];
        result[threadIdx.x] = 0;
        for (int i = start + lane; i < end; i+= WARP_SIZE) {
            result[threadIdx.x] += values[i] * x[col_ind[i]];
        }
        // Threads in a warp are synchronized, so we can do this
        int half = WARP_SIZE / 2;
        while (half > 0) {
            if (lane < half) {
                result[threadIdx.x] += result[threadIdx.x + half];
            }
            half /= 2;
        }
        if (lane == 0) {
            y[r] = result[threadIdx.x];
        }
    }
}

/**
 * @brief   Caller function for vector dot product in CUDA.
 *
 * @param   v1  The first vector.
 * @param   v2  The second vector.
 *
 * @return  The result of dot product of v1 and v2.
 */
template <typename T>
T cuda_dot_product(const vector<T> &v1, const vector<T> &v2) {
    int n = v1.size();
    assert(n == v2.size());
    const int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Malloc device space
    T *x, *y, *z;
    hipMalloc(&x, sizeof(T) * n);
    hipMalloc(&y, sizeof(T) * n);
    hipMalloc(&z, sizeof(T) * blocks);

    // Transfer data from host to device
    hipMemcpy(x, v1.data(), sizeof(T) * n, hipMemcpyHostToDevice);
    hipMemcpy(y, v2.data(), sizeof(T) * n, hipMemcpyHostToDevice);

    // Run kernel
    dot_product_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(n, x, y, z);
    hipDeviceSynchronize();

    // Transfer result back from device to host
    T z_host[blocks];
    T result(0);
    hipMemcpy(z_host, z, sizeof(T) * blocks, hipMemcpyDeviceToHost);
    for (int i = 0; i < blocks; i++) {
        result += z_host[i];
    }

    // Release device space
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return result;
}

/**
 * @brief   Caller function for inplace vector multiplication in CUDA.
 *
 * @param   v   The vector to multiply to.
 * @param   k   The value to multiply.
 */
template <typename T>
void cuda_multiply_inplace(vector<T> &v, const T &k) {
    int n = v.size();

    // Malloc device space
    T *x;
    hipMalloc(&x, sizeof(T) * n);

    // Transfer data from host to device
    hipMemcpy(x, v.data(), sizeof(T) * n, hipMemcpyHostToDevice);

    // Run kernel
    const int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    multiply_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(n, x, k);
    hipDeviceSynchronize();

    // Transfer result back from device to host
    hipMemcpy(v.data(), x, sizeof(T) * n, hipMemcpyDeviceToHost);

    // Release device space
    hipFree(x);
}

/**
 * @brief   Caller function for inplace vector add in CUDA.
 *
 * @param   v   The vector to add to.
 * @param   k   The value to add.
 */
template <typename T>
void cuda_add_inplace(vector<T> &v, const T &k) {
    int n = v.size();

    // Malloc device space
    T *x;
    hipMalloc(&x, sizeof(T) * n);

    // Transfer data from host to device
    hipMemcpy(x, v.data(), sizeof(T) * n, hipMemcpyHostToDevice);

    // Run kernel
    const int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(n, x, k);
    hipDeviceSynchronize();

    // Transfer result back from device to host
    hipMemcpy(v.data(), x, sizeof(T) * n, hipMemcpyDeviceToHost);

    // Release device space
    hipFree(x);
}

/**
 * @brief   Caller function for inplace vector saxpy (y += a * x) in CUDA.
 *
 * @param   y   The vector to add to.
 * @param   x   The vector to multiply to.
 * @param   a   The value to multiply.
 */
template <typename T>
void cuda_saxpy_inplace(vector<T> &y, const T &a, const vector<T> &x) {
    int n = y.size();
    assert(n = x.size());

    // Malloc device space
    T *dev_y;
    T *dev_x;
    hipMalloc(&dev_y, sizeof(T) * n);
    hipMalloc(&dev_x, sizeof(T) * n);

    // Transfer data from host to device
    hipMemcpy(dev_y, y.data(), sizeof(T) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x.data(), sizeof(T) * n, hipMemcpyHostToDevice);

    // Run kernel
    const int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    saxpy_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(n, dev_y, dev_x,
        a);
    hipDeviceSynchronize();

    // Transfer result back from device to host
    hipMemcpy(y.data(), dev_y, sizeof(T) * n, hipMemcpyDeviceToHost);

    // Release device space
    hipFree(dev_y);
    hipFree(dev_x);
}

/**
 * @brief   Caller function for vector l2 norm in CUDA.
 *
 * @param   v  The vector.
 *
 * @return  The l2 norm of the vector v.
 */
template <typename T>
T cuda_l2_norm(const vector<T> &v) {
    int n = v.size();
    const int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Malloc device space
    T *x, *z;
    hipMalloc(&x, sizeof(T) * n);
    hipMalloc(&z, sizeof(T) * blocks);

    // Transfer data from host to device
    hipMemcpy(x, v.data(), sizeof(T) * n, hipMemcpyHostToDevice);

    // Run kernel
    dot_product_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(n, x, x, z);
    hipDeviceSynchronize();

    // Transfer result back from device to host
    T z_host[blocks];
    T result(0);
    hipMemcpy(z_host, z, sizeof(T) * blocks, hipMemcpyDeviceToHost);
    for (int i = 0; i < blocks; i++) {
        result += z_host[i];
    }

    // Release device space
    hipFree(x);
    hipFree(z);

    return T(sqrt(result));
}

/**
 * @brief   Caller function for naive sparse matrix multiplication in CUDA.
 *
 * @param   m   The matrix to multiply.
 * @param   v   The vector to multiply.
 *
 * @return  The result of matrix vector multiplication of m*v.
 */
template <typename T>
vector<T> cuda_naive_multiply(const csr_matrix<T> &m, const vector<T> &v) {
    int rows = m.row_size();
    int cols = m.col_size();
    int nonzeros = m.nonzeros();
    assert(cols == v.size());

    // Malloc device space
    int *row_ptr, *col_ind;
    T *values, *x, *y;
    hipMalloc(&row_ptr, sizeof(int) * (rows + 1));
    hipMalloc(&col_ind, sizeof(int) * nonzeros);
    hipMalloc(&values, sizeof(T) * nonzeros);
    hipMalloc(&x, sizeof(T) * cols);
    hipMalloc(&y, sizeof(T) * cols);

    // Transfer data from host to device
    hipMemcpy(row_ptr, m.row_ptr_data(), sizeof(int) * (rows + 1),
        hipMemcpyHostToDevice);
    hipMemcpy(col_ind, m.col_ind_data(), sizeof(int) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(values, m.values_data(), sizeof(T) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(x, v.data(), sizeof(T) * cols, hipMemcpyHostToDevice);

    // Run kernel
    const int blocks = (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    double start_time = cycle_timer::current_seconds();
    naive_multiply_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows, row_ptr,
        col_ind, values, x, y);
    hipDeviceSynchronize();
    double end_time = cycle_timer::current_seconds();
    printf("gpu naive multiply kernel: %f\n", end_time - start_time);

    // Transfer result back from device to host
    vector<T> result(cols);
    hipMemcpy(result.data(), y, sizeof(T) * cols, hipMemcpyDeviceToHost);

    // Release device space
    hipFree(row_ptr);
    hipFree(col_ind);
    hipFree(values);
    hipFree(x);
    hipFree(y);

    return result;
}

/**
 * @brief   Caller function for warp sparse matrix multiplication in CUDA.
 *
 * @param   m   The matrix to multiply.
 * @param   v   The vector to multiply.
 *
 * @return  The result of matrix vector multiplication of m*v.
 */
template <typename T>
vector<T> cuda_warp_multiply(const csr_matrix<T> &m, const vector<T> &v) {
    int rows = m.row_size();
    int cols = m.col_size();
    int nonzeros = m.nonzeros();
    assert(cols == v.size());

    // Malloc device space
    int *row_ptr, *col_ind;
    T *values, *x, *y;
    hipMalloc(&row_ptr, sizeof(int) * (rows + 1));
    hipMalloc(&col_ind, sizeof(int) * nonzeros);
    hipMalloc(&values, sizeof(T) * nonzeros);
    hipMalloc(&x, sizeof(T) * cols);
    hipMalloc(&y, sizeof(T) * cols);

    // Transfer data from host to device
    hipMemcpy(row_ptr, m.row_ptr_data(), sizeof(int) * (rows + 1),
        hipMemcpyHostToDevice);
    hipMemcpy(col_ind, m.col_ind_data(), sizeof(int) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(values, m.values_data(), sizeof(T) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(x, v.data(), sizeof(T) * cols, hipMemcpyHostToDevice);

    // Run kernel
    const int warps_per_block = THREADS_PER_BLOCK / WARP_SIZE;
    const int blocks = (rows + warps_per_block - 1) / warps_per_block;
    double start_time = cycle_timer::current_seconds();
    warp_multiply_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows, row_ptr,
        col_ind, values, x, y);
    hipDeviceSynchronize();
    double end_time = cycle_timer::current_seconds();
    printf("gpu warp multiply kernel: %f\n", end_time - start_time);

    // Transfer result back from device to host
    vector<T> result(cols);
    hipMemcpy(result.data(), y, sizeof(T) * cols, hipMemcpyDeviceToHost);

    // Release device space
    hipFree(row_ptr);
    hipFree(col_ind);
    hipFree(values);
    hipFree(x);
    hipFree(y);

    return result;
}

/**
 * @brief   Caller function for naive Lanczos algorithm in CUDA.
 *
 * @param   m   The matrix to do operations on.
 * @param   v   The initial vector with norm 1.
 * @param   k   The iteration times for lanczos algorithm.
 *
 * @return  The tridiagonal matrix result of lanczos algorithm.
 */
template <typename T>
symm_tridiag_matrix<T> cuda_naive_lanczos(const csr_matrix<T> &m,
    const vector<T> &v, const int k) {
    // TODO: This function is untested
    symm_tridiag_matrix<T> result(k);

    int rows = m.row_size();
    int cols = m.col_size();
    int nonzeros = m.nonzeros();
    const int blocks = (rows+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK;
    assert(rows == cols);
    assert(cols == v.size());

    // Malloc device space
    int *row_ptr, *col_ind;
    T *values, *x, *x_prev, *x_tmp, *y, *z;
    hipMalloc(&row_ptr, sizeof(int) * (rows + 1));
    hipMalloc(&col_ind, sizeof(int) * nonzeros);
    hipMalloc(&values, sizeof(T) * nonzeros);
    hipMalloc(&x, sizeof(T) * cols);
    hipMalloc(&x_prev, sizeof(T) * cols);
    hipMalloc(&y, sizeof(T) * cols);
    hipMalloc(&z, sizeof(T) * blocks);

    // Host space
    T z_host[blocks];
    T sum(0);

    // Transfer data from host to device
    hipMemcpy(row_ptr, m.row_ptr_data(), sizeof(int) * (rows + 1),
        hipMemcpyHostToDevice);
    hipMemcpy(col_ind, m.col_ind_data(), sizeof(int) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(values, m.values_data(), sizeof(T) * nonzeros,
        hipMemcpyHostToDevice);
    hipMemcpy(x, v.data(), sizeof(T) * cols, hipMemcpyHostToDevice);

    // Run kernel
    for (int i = 0; i < k; i++) {
        // y_i = M*x_i
        naive_multiply_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,
            row_ptr, col_ind, values, x, y);
        hipDeviceSynchronize();
        // alpha_i <- y_i*x_i
        dot_product_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,x,y,z);
        hipDeviceSynchronize();
        hipMemcpy(z_host, z, sizeof(T) * blocks, hipMemcpyDeviceToHost);
        sum = 0;
        for (int j = 0; j < blocks; j++) {
            sum += z_host[i];
        }
        result.alpha(i) = sum;
        // y_i <- y_i - alpha_i*x_i - beta_i*x_(i-1)
        if (i == 0) {
            saxpy_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,
                y, x, -sum);
            hipDeviceSynchronize();
            x_tmp = x;
            x = x_prev;
            x_prev = x_tmp;
        } else {
            saxpy_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,
                y, x, -sum);
            hipDeviceSynchronize();
            saxpy_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,
                y, x_prev, -result.beta(i-1));
            hipDeviceSynchronize();
            x_tmp = x;
            x = x_prev;
            x_prev = x_tmp;
        }
        // beta_(i+1) <- ||y_i||
        dot_product_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows,y,y,z);
        hipDeviceSynchronize();
        hipMemcpy(z_host, z, sizeof(T) * blocks, hipMemcpyDeviceToHost);
        sum = 0;
        for (int j = 0; j < blocks; j++) {
            sum += z_host[i];
        }
        result.beta(i) = T(sqrt(sum));
        // x_(i+1) <- y_i / beta_(i+1)
        multiply_inplace_kernel<T><<<blocks, THREADS_PER_BLOCK>>>(rows, y,
            1/result.beta(i));
        hipDeviceSynchronize();
        x_tmp = x;
        x = y;
        y = x_tmp;
    }

    // Release device space
    hipFree(row_ptr);
    hipFree(col_ind);
    hipFree(values);
    hipFree(x);
    hipFree(x_prev);
    hipFree(y);
    hipFree(z);

    return result;
}

template __global__ void dot_product_kernel<float>(const int,
    const float *, const float *, float *);
template float cuda_dot_product<float>(const vector<float> &v1,
    const vector<float> &v2);
template __global__ void multiply_inplace_kernel<float>(const int, float *,
    const float);
template void cuda_multiply_inplace<float>(vector<float> &v, const float &k);
template __global__ void add_inplace_kernel<float>(const int, float *,
    const float);
template void cuda_add_inplace<float>(vector<float> &v, const float &k);
template __global__ void saxpy_inplace_kernel<float>(const int, float *,
    const float *, const float);
template void cuda_saxpy_inplace<float>(vector<float> &y, const float &a,
    const vector<float> &x);
template float cuda_l2_norm(const vector<float> &v);

template __global__ void naive_multiply_kernel(const int, const int *,
    const int *, const float *, const float *, float *);
template vector<float> cuda_naive_multiply(const csr_matrix<float> &m,
    const vector<float> &v);
template __global__ void warp_multiply_kernel(const int, const int *,
    const int *, const float *, const float *, float *);
template vector<float> cuda_warp_multiply(const csr_matrix<float> &m,
    const vector<float> &v);
